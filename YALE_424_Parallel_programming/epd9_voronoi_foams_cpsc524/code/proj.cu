#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>


#include "voronoi_query.cpp"

// Error checking macro from here: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define CUDA_ERR_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line) {
	if(code != hipSuccess) {
		fprintf(stderr, "GPU error: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

__host__ __device__ FP rho(point q) { return (FP) 0.5; }

__global__ void gpu_foam(VOX *foam, dim3 shape, int z) {
	float step = 0.05;

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	point q;
	q.x = step*x;
	q.y = step*y;
	q.z = step*z;

  if(x < shape.x && y < shape.y) {
  	VOX result = voronoi_query(q);

  	if(sizeof(VOX)*(x + y*shape.x) >= shape.x*shape.y * sizeof(VOX)) {
  		printf("How did you get here?\n"); // TODO remove
  	}

  	foam[x + y*shape.x] = result;
  }
}

int main(int argc, char *argv[]) {

	int i, j;
	dim3 shape;
	shape.x = 100;
	shape.y = 100;
	shape.z = 100;
	dim3 Grid(10, 10, 1); //Grid structure
	dim3 Block(10, 10, 1); //Block structure


	size_t size = shape.x * shape.y *sizeof(VOX);
	VOX  *dev_foam, *foam;
	foam = (VOX *) malloc(size);
	CUDA_ERR_CHK(hipMalloc((void**)&dev_foam, size));

	for(int z = 0; z < shape.z; z++) {
		gpu_foam<<<Grid, Block>>>(dev_foam, shape, z);
		// CUDA_ERR_CHK(hipMemset(dev_foam, 255, size)); // This works instead of gpu_foam<<...>>>

		CUDA_ERR_CHK(hipGetLastError());
		CUDA_ERR_CHK(hipMemcpy(foam, dev_foam, size, hipMemcpyDeviceToHost));
		
		printf("Z layer %d:\n", z);
		for(i=0; i<shape.y; i++) {
			for(j=0; j<shape.x; j++) {
				printf("%c", foam[shape.x*i+j] == 0 ? ' ' : '#');
			}
			printf("\n");
		}
	}

	
	free(foam);
	CUDA_ERR_CHK(hipFree(dev_foam));

	return 0;
}